#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <cstdlib>
#include <iostream>
#include <cmath>
#include <chrono>
using namespace std;

struct square
{
    __host__ __device__
    double operator()(const double& x) const {
        return x * x;
    }
};

double magnitude_gpu(thrust::device_vector<double>& v) {
    double sum_of_squares = thrust::transform_reduce(v.begin(), v.end(), square(), 0.0f, thrust::plus<double>());
    return std::sqrt(sum_of_squares);
}

double magnitude_cpu(const std::vector<double>& v) {
    double sum_of_squares = 0.0f;
    for (size_t i = 0; i < v.size(); ++i) {
        sum_of_squares += v[i] * v[i];
    }
    return std::sqrt(sum_of_squares);
}

int main() {
    const int N = 1000000;  // Tamanho do vetor
    thrust::host_vector<double> h_v(N);

    for (int i = 0; i < N; ++i) { h_v[i] = static_cast<double>(rand() % 10); }

    thrust::device_vector<double> d_v = h_v;

    // Conversão para std::vector para a implementação na CPU
    std::vector<double> v_cpu(h_v.begin(), h_v.end());

    auto start_gpu = chrono::high_resolution_clock::now();
    double result_gpu = magnitude_gpu(d_v);
    auto end_gpu = chrono::high_resolution_clock::now();
    chrono::duration<double> duration_gpu = end_gpu - start_gpu;

    auto start_cpu = chrono::high_resolution_clock::now();
    double result_cpu = magnitude_cpu(v_cpu);
    auto end_cpu = chrono::high_resolution_clock::now();
    chrono::duration<double> duration_cpu = end_cpu - start_cpu;

    cout << "Magnitude GPU: " << result_gpu << endl;
    cout << "Tempo GPU: " << duration_gpu.count() << " segundos" << endl;

    cout << "Magnitude CPU: " << result_cpu << endl;
    cout << "Tempo CPU: " << duration_cpu.count() << " segundos" << endl;

    return 0;
}
